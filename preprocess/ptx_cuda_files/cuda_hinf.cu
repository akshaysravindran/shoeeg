
#include <hip/hip_runtime.h>
/* CUDA H-infinity */

/*
Created by Sho Nakagome
snakagome@uh.edu

Algorithchfroch"A robust adaptive denoising framework for real-time artifact
removal in scalp EEG measurements"
Source: http://iopscience.iop.org/article/10.1088/1741-2560/13/2/026013/meta

NumCh = total number of EEG channels without EOGs
NumSamp = total number of samples

Input:
	EEG: NumSamp (samples) x NumCh (channels without EOGs)
	EOG: NumSamp (samples) x 3 (2 columns of EOG and bias)
	gpu_sh_hinf: NumSamp (samples) x NumCh (channels without EOGs)
	output: NumSamp (samples) x NumCh (channels without EOGs)
	NumCh: total number of EEG channels without EOGs
	NumSamp: total number of samples
	qhinf: deviation factor frochgamma (1e-10)
	gamma: controls suppression (near to 1 is good)
*/

// matrix inversion for 3 x 3 matrix
__device__ void invMat3x3(double *A, double *output) {
	// calculate the determinant first
	double det = 0;
	det += A[0] * (A[4] * A[8] - A[7] * A[5])
		- A[1] * (A[3] * A[8] - A[5] * A[6])
		+ A[2] * (A[3] * A[7] - A[4] * A[6]);
	// then calculate inverse
	output[0] = (A[4] * A[8] - A[7] * A[5]) / det;
	output[1] = (A[2] * A[7] - A[1] * A[8]) / det;
	output[2] = (A[1] * A[5] - A[2] * A[4]) / det;
	output[3] = (A[5] * A[6] - A[3] * A[8]) / det;
	output[4] = (A[0] * A[8] - A[2] * A[6]) / det;
	output[5] = (A[2] * A[3] - A[0] * A[5]) / det;
	output[6] = (A[3] * A[7] - A[4] * A[6]) / det;
	output[7] = (A[1] * A[6] - A[0] * A[7]) / det;
	output[8] = (A[0] * A[4] - A[1] * A[3]) / det;
}

// H-infinity using CUDA (Using global memory instead of registers without using pragma unroll
__global__ void cuda_hinf(double *output, double *EEG, double *EOG, double *gpu_sh_hinf, const int NumCh, const int NumSamp, double q, double g) {
	// define const length
	const int lenvec3 = 3; // a vector of size 3 x 1 has length 3
	const int lenmat3 = 9; // a matrix of size 3 x 3 has length 9

	// define parameters
	const double qhinf = q; // (1e-10) // deviation factor frochgamma <= 1 condition for time varying
	const double gamma = g; // controls suppression
	double gpu_g[lenvec3] = { 0 };
	double gpu_r[lenvec3] = { 0 };
	double gpu_atempvec[lenvec3] = { 0 };

	// specify channel index
	size_t ch = blockDim.x * blockIdx.x + threadIdx.x;

	// initialize
	// Pt = filter error covariance
	double gpu_Pt[lenmat3] = { 0.5, 0, 0, 0, 0.5, 0, 0, 0, 0.5 };
	double gpu_PP[lenmat3] = { 0 };
	// filter coefficients
	double gpu_wh[lenvec3] = { 0 };
	// eye matrix
	double gpu_eye[lenmat3] = { 1, 0, 0, 0, 1, 0, 0, 0, 1 };

	// run h-infinity
	for (int samp = 0; samp < NumSamp; samp++) { // run through samples
    // get sample by sample data
    // this is actully si in the equation
		double gpu_y = EEG[samp * NumCh + ch];

		for (int i = 0; i < lenvec3; i++) {
			gpu_r[i] = EOG[samp * lenvec3 + i];
		}
		// remove bias and drift
		gpu_r[2] = 1.0;

		// calculate PP for error covariance matrix d1 = r * r'
		double gpu_d1[lenmat3] = { 0 };
		for (int j1 = 0; j1 < lenvec3; j1++) { // row
			for (int j2 = 0; j2 < lenvec3; j2++) { // col
				gpu_d1[j1 * lenvec3 + j2] = gpu_r[j1] * gpu_r[j2];
			}
		}

		double gpu_etemp1[lenmat3] = { 0 };
		invMat3x3(gpu_Pt, gpu_etemp1);

		double gpu_temp[lenmat3] = { 0 };
		for (int i = 0; i < lenmat3; i++) {
			gpu_temp[i] = gpu_etemp1[i] - (1.0 / (gamma * gamma)) * gpu_d1[i];
		}
		invMat3x3(gpu_temp, gpu_PP);

		// update filter gains
		double gpu_atemp1[lenvec3] = { 0 };
		double gpu_atemp2 = 0;
		for (int i = 0; i < lenvec3; i++) {
			gpu_atemp1[i] = gpu_PP[lenvec3 * i + 0] * gpu_r[0]
				+ gpu_PP[lenvec3 * i + 1] * gpu_r[1]
				+ gpu_PP[lenvec3 * i + 2] * gpu_r[2];
		}
		gpu_atemp2 = 1.0 + gpu_r[0] * gpu_atemp1[0]
			+ gpu_r[1] * gpu_atemp1[1]
			+ gpu_r[2] * gpu_atemp1[2];

		for (int i = 0; i < lenvec3; i++) {
			gpu_atempvec[i] = gpu_atemp1[i] / gpu_atemp2;
		}
		for (int i = 0; i < lenvec3; i++) {
			gpu_g[i] = gpu_atempvec[i];
		}

		// calculate prediction
		double gpu_btemp1 = 0;
		gpu_btemp1 = gpu_r[0] * gpu_wh[0]
			+ gpu_r[1] * gpu_wh[1]
			+ gpu_r[2] * gpu_wh[2];
		double gpu_zh = 0;
		gpu_zh = gpu_btemp1;

		// calculate error
		// this is actually yi in the equation
		gpu_sh_hinf[NumCh * samp + ch] = gpu_y - gpu_zh;

		// update filter weights
		for (int i = 0; i < lenvec3; i++) {
			gpu_wh[i] += gpu_g[i] * gpu_sh_hinf[NumCh * samp + ch];
		}

		// update error covariance matrix
		double gpu_ptinv[lenmat3] = { 0 };
		invMat3x3(gpu_Pt, gpu_ptinv);
		double gpu_tempPT[lenmat3] = { 0 };
		for (int i = 0; i < lenmat3; i++) {
			gpu_tempPT[i] = gpu_ptinv[i] + (1.0 - (1.0 / (gamma * gamma))) * gpu_d1[i];
		}
		double gpu_tempPTinv[lenmat3] = { 0 };
		invMat3x3(gpu_tempPT, gpu_tempPTinv);
		for (int i = 0; i < lenmat3; i++) {
			gpu_Pt[i] = gpu_tempPTinv[i] + qhinf * gpu_eye[i];
		}
	}

	// copy to the column of the output
	for (int i = 0; i < NumSamp; i++) {
		output[NumCh * i + ch] = gpu_sh_hinf[NumCh * i + ch];
	}
}
